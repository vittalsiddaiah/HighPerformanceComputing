#include "hip/hip_runtime.h"
#include <stdio.h>
#include <fcntl.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <iostream>
#include "StdHdr.hpp"


#define CUDA_TILE_SIZE 64
#define GridSize(matrixSize) (uint32_t)ceil((float)(matrixSize)/(float)CUDA_TILE_SIZE)

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// CUDA PROGRAMMING: Kernel Function
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void DeviceMatrixMultiply(float *ptrLhs, float *ptrRhs, float *ptrAvg, int matrixSize)  {
    //CUDA Tile Declarations
    __shared__ float lhs[CUDA_TILE_SIZE][CUDA_TILE_SIZE];
    __shared__ float rhs[CUDA_TILE_SIZE][CUDA_TILE_SIZE];

    //Initialized the allocated block to Zero
    //TODO Need to check if this step is required as we are copying the data from the source...
    lhs[threadIdx.y][threadIdx.x] = 0.0;
    rhs[threadIdx.y][threadIdx.x] = 0.0;

    // ids of thread
    int Row = blockDim.y*blockIdx.y + threadIdx.y;
    int Col = blockDim.x*blockIdx.x + threadIdx.x;
    float product = 0.0;


    for (int k = 0; k < (((matrixSize - 1)/ CUDA_TILE_SIZE) + 1); k++)  {
        if ((Row < matrixSize) && (threadIdx.x + (k*CUDA_TILE_SIZE)) < matrixSize)
            lhs[threadIdx.y][threadIdx.x] = ptrLhs[(Row*matrixSize) + threadIdx.x + (k*CUDA_TILE_SIZE)];
        else
            lhs[threadIdx.y][threadIdx.x] = 0.0; // if outside the boundaries of the matrix

        if ((Col < matrixSize) && (threadIdx.y + k*CUDA_TILE_SIZE) < matrixSize)
            rhs[threadIdx.y][threadIdx.x] = ptrRhs[(threadIdx.y + k*CUDA_TILE_SIZE)*matrixSize + Col];
        else
            rhs[threadIdx.y][threadIdx.x] = 0.0; //if outside the boundary of the matrix

        // making sure all threads have arrived before multiplications
        __syncthreads();

        // multiplying elements present in the current tile
        for (int j = 0; j < CUDA_TILE_SIZE; ++j)
            product += lhs[threadIdx.y][j] * rhs[j][threadIdx.x];  //Product of each thread

    }

    // Saving final result to Matrix C
    if (Row < matrixSize && Col < matrixSize) {
        ptrAvg[Row*matrixSize + Col] = product;
    }
}


////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// Host side code (Compiled for General Purpose
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
int main(int argc, char ** argv) {
    std::cout << Label() << std::endl;
    uint64_t beginTotTime, endTotTime;
    GetCurrentTimeUS(beginTotTime);

    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    // Process input arguments.
    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    int matrixSize = atoi(argv[1]);
    int iterations = atoi(argv[2]);
    uint64_t totalSize = matrixSize*matrixSize;
    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////


    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    // Host Side Memory Allocations.
    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    float *hostLhs;
    float *hostRhs;
    float *hostAvg;
    hostLhs = new float[totalSize];
    hostRhs = new float[totalSize];
    hostAvg = new float[totalSize];
    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    // Device/GPU Side Memory Allocations.
    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    float *deviceLhs;
    float *deviceRhs;
    float *deviceAvg;
    hipMalloc((void **)&deviceLhs, sizeof(float)*totalSize);
    hipMalloc((void **)&deviceRhs, sizeof(float)*totalSize);
    hipMalloc((void **)&deviceAvg, sizeof(float)*totalSize);
    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////


    std::cout << "  HostRand,AvgHostRand,   MemcpH2C,AvgMemcpH2C,  MMPlyCuda,AvgMMPlyCuda,MinMMPlyCuda,  MemcpC2H,AvgMemcpC2H" << std::endl << std::flush;
    double avgRandTime = 0.0, avgMemCpH2C = 0.0, avgMmply = 0.0, avgMemCpC2H = 0.0, minMmply = 999.0;
    for(size_t iteration = 1; iteration < iterations+1; iteration++)  {
        ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
        // Host size vector randomization
        ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
        //OMP Parallel if required here...
        uint64_t beginRndTime, endRndTime;
        GetCurrentTimeUS(beginRndTime);
        for(size_t offset=0; offset < totalSize; offset++)
            hostLhs[offset]  = Rand, hostRhs[offset]  = Rand;
        GetCurrentTimeUS(endRndTime);
        double randTimeSecs = NanoSecs2Secs(endRndTime - beginRndTime);
        //std::cout << "Randomization Completed in     : " << randTimeSecs << "Secs. " << std::endl << std::flush;
        avgRandTime += randTimeSecs;
        std::cout << FAvg_f << randTimeSecs << ", " << FAvg_f << (avgRandTime/iteration) << ", " << std::flush;
        ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

        // Memory Copy From CPU addressable memory to GPU addressable memory
        ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
        uint64_t beginCmcpyTime, endCmcpyTime;
        GetCurrentTimeUS(beginCmcpyTime);
        hipMemcpy(deviceLhs, hostLhs, sizeof(float)*totalSize, hipMemcpyHostToDevice);
        hipMemcpy(deviceRhs, hostRhs, sizeof(float)*totalSize, hipMemcpyHostToDevice);
        GetCurrentTimeUS(endCmcpyTime);
        double cudaMemCpyTimeSecs = NanoSecs2Secs(endCmcpyTime - beginCmcpyTime);
        avgMemCpH2C += cudaMemCpyTimeSecs;
        //std::cout << "Host 2 Cuda Memcpy Completed in     : " << cudaMemCpyTimeSecs << "Secs. " << std::endl << std::flush;
        std::cout << FAvg_f << cudaMemCpyTimeSecs << ", " << FAvg_f << (avgMemCpH2C/iteration) << ", " << std::flush;
        ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

        ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
        // Defining CUDA topology/configuration, Initializing the Grid and Block dimensions
        // (dim3 is an integer vector type that can be used in CUDA code.)
        ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
        //uint32_t gridSize = (uint32_t)ceil((float)matrixSize/(float)CUDA_TILE_SIZE);
        dim3 dimGrid(GridSize(matrixSize), GridSize(matrixSize), 1);
        dim3 dimBlock(CUDA_TILE_SIZE, CUDA_TILE_SIZE, 1);

        uint64_t beginMpyTime, endMpyTime;
        GetCurrentTimeUS(beginMpyTime);
        DeviceMatrixMultiply<<<dimGrid, dimBlock>>>(deviceLhs, deviceRhs, deviceAvg, matrixSize);
        GetCurrentTimeUS(endMpyTime);
        double mplyEndTimeSecs = NanoSecs2Secs(endMpyTime - beginMpyTime);
        avgMmply += mplyEndTimeSecs;
        minMmply = (mplyEndTimeSecs < minMmply) ? mplyEndTimeSecs : minMmply;
        //std::cout << "CUDA Multiplication Completed in :" << mplyEndTimeSecs << "Secs. "<< std::endl << std::flush;
        std::cout << FAvg_f << mplyEndTimeSecs << ", " << FAvg_f << (avgMmply/iteration) << ", " << FAvg_f << minMmply << ", "<< std::flush;
        ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

        // Another way to sync theads from the GPU
        hipDeviceSynchronize();

        ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
        // Memory Copy From GPU memory to CPU addressable memory
        ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
        uint64_t beginCMcpyTime, endCMcpyTime;
        GetCurrentTimeUS(beginCMcpyTime);
        hipMemcpy(hostRhs, deviceAvg, sizeof(float)*totalSize, hipMemcpyDeviceToHost);
        for(size_t offset=0; offset < totalSize; offset++) hostAvg[offset] += hostRhs[offset];
        GetCurrentTimeUS(endCMcpyTime);
        double cudaMEMCpyTimeSecs = NanoSecs2Secs(endCMcpyTime - beginCMcpyTime);
        avgMemCpC2H += cudaMEMCpyTimeSecs;
        //std::cout << "Cuda 2 Host Memcpy Completed in     : " << cudaMEMCpyTimeSecs << "Secs. " << std::endl << std::flush;
        std::cout << FAvg_f << cudaMEMCpyTimeSecs << ", " << FAvg_f << (avgMemCpC2H/iteration) << ", " << std::endl << std::flush;
        ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    }

    for(size_t offset=0; offset < totalSize; offset++) hostAvg[offset] /=iterations;
    int fd_avgFile = open("CudaAverageMatrixFile.dat",O_CREAT|O_WRONLY,0775);
    int bytesWritten;
    if(fd_avgFile >0)   {
        if ((bytesWritten = write(fd_avgFile, hostAvg, sizeof(*hostAvg)*totalSize)) == -1)
            std::cerr << "Error in writing to the file ..." << bytesWritten << " Bytes written" << std::endl << std::flush;
        close(fd_avgFile);
     }

    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    // Memory De-Allocation of Cuda target / Device Memory
    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    if (NULL != deviceLhs) hipFree(deviceLhs);
    if (NULL != deviceRhs) hipFree(deviceRhs);
    if (NULL != deviceAvg) hipFree(deviceAvg);
    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    // Memory De-Allocation of host Memory
    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    if (NULL != hostLhs) delete[]hostLhs;
    if (NULL != hostRhs) delete[]hostRhs;
    if (NULL != hostAvg) delete[]hostAvg;
    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    std::cout << "Program Execution Completed " << std::endl << std::flush;
    GetCurrentTimeUS(endTotTime);
    double totTimeSecs = NanoSecs2Secs(endTotTime - beginTotTime);

    char hostName[64];
    char userName[64];
    gethostname(hostName, 64);
    getlogin_r(userName, 64);

    std::stringstream sstr;
    sstr << "Cuda_Score_Card_" << TimeStamp() << ".txt";
    std::ofstream rptTxt_f;
    rptTxt_f.open(sstr.str().c_str());
    rptTxt_f << "-----------------   Cuda Matrix Multiplication Score Card -----------------" << std::endl;
    rptTxt_f << "                        Matrix Size : " << matrixSize << " x " << matrixSize << std::endl;
    rptTxt_f << "                         Iterations : " << iterations << std::endl;
    rptTxt_f << "                          Host Name : " << hostName << std::endl;
    rptTxt_f << "                          User Name : " << userName << std::endl;
    rptTxt_f << "    Average Random Number Fill Time : " << FAvg_f << avgRandTime/iterations  << " secs." << std::endl;
    rptTxt_f << "     Average Host to Cuda Copy Time : " << FAvg_f << avgMemCpH2C/iterations  << " secs." << std::endl;
    rptTxt_f << "     Average Cuda to Host Copy Time : " << FAvg_f << avgMemCpC2H/iterations  << " secs." << std::endl;
    rptTxt_f << "       Average Matrix Multiply Time : " << FAvg_f << avgMmply/iterations << " secs." << std::endl;
    rptTxt_f << "                 Best Multiply Time : " << FAvg_f << minMmply << " secs." << std::endl;
    rptTxt_f << "      Total Time for all Iterations : " << FAvg_f << totTimeSecs << " secs." << std::endl;
    rptTxt_f << "---------------------------------------------------------------------------" << std::endl;
    rptTxt_f.close();
    return 0;
}

